#include "hip/hip_runtime.h"
/*
 * Performs WOLA FFT to channelize data. GPU/CPU version.
 *
 * gpuWola(y,f_tap,N,Dec)
 * Inputs: signal, filter tap coeffs, number of FFT bins, decimation factor
 *
 * NOTE: For some reason, compiling GPU code requires explicit specification of 
 * CUDA libraries location with -L, even if environment variable is set.
*/

#include "mex.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <ipp.h>

#include <windows.h>
#include <process.h>

// #include <pthread.h>

#include <hip/hip_runtime.h>
#include <npp.h>

#define NUM_THREADS 24

// timing functions
double PCFreq = 0.0;
__int64 CounterStart = 0;
int StartCounter()
{
    LARGE_INTEGER li;
    if(!QueryPerformanceFrequency(&li))
    printf("QueryPerformanceFrequency failed!\n");

    PCFreq = ((double)li.QuadPart)/1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
	return (int)CounterStart;
}

int GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return (int)li.QuadPart;
}

// definition of thread data
struct thread_data{
	int thread_t_ID;
	int thread_L;
	int thread_N;
	int thread_Dec;
	int thread_nprime_total;
	Npp16sc *thread_h_rawdata;
	Npp16sc *thread_d_in;
	Npp16sc *thread_d_in_next;
	Npp32fc *thread_d_out;
	Npp32fc *thread_d_out_next;
	Npp32f *thread_d_ftap;
	Npp32fc *thread_h_out;
	hipStream_t thread_stream;
	hipStream_t thread_copystream;
	int *thread_nprime_startIdx;
	
	Ipp8u *thread_pDFTBuffer;
	IppsDFTSpec_C_32fc *thread_pDFTSpec;

	Ipp32fc *thread_out; // for R2018
};

// declare global thread stuff
struct thread_data thread_data_array[NUM_THREADS];


// the wola kernel doing the elementwise product
__global__
void wola_front(int N, int L, Npp16sc *d_in, Npp32fc *d_out, Npp32f *d_ftap)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x; 
	
	Npp32f re, im; // oh snap writing to stack first almost halves the kernel time lol
	
	for (int a = index; a<N; a+=stride){ // probably just launch total N threads, calculate blockDim required
		re = 0;
		im = 0;
		
		for (int b = 0; b < L/N; b++){
			re = re + (Npp32f)(d_in[L-1 - (b*N+a)].re) * d_ftap[b*N+a];
			im = im + (Npp32f)(d_in[L-1 - (b*N+a)].im) * d_ftap[b*N+a];
		}
		
		d_out[a].re = re;
		d_out[a].im = im;
	}
}

// this works but sometimes throws errors, maybe don't use it..
__global__ void device_copy_vector4_int16_kernel(Npp16sc *d_out, Npp16sc *d_in, int len){ // let's see if this is faster than memcpyAsyncs
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = idx; i < len/4; i += stride){
		reinterpret_cast<int4*>(d_out)[i] = reinterpret_cast<int4*>(d_in)[i]; // you're transferring 4 ints at a time, this is the same as 4 Npp16sc since the real and imag components are equal to 32 bits (one int)
	}
	
	// in one thread, process final elements
	int remainder = len%4;
	if (idx==len/4 && remainder!=0){
		while(remainder){
			idx = len - remainder--;
			d_out[idx] = d_in[idx];
		}
	}
} // you might want to check if this is really copying correctly

// if you don't have this the mexfile will crash after the second execution because lul
__host__ void cleanUp(){
	hipDeviceReset();
}

unsigned __stdcall threaded_wola(void *pArgs){
// void *threaded_wola(void *pArgs){
	struct thread_data *inner_data;
	inner_data = (struct thread_data *)pArgs;
	
	int t_ID = inner_data->thread_t_ID;
	int L = inner_data->thread_L;
	int N = inner_data->thread_N;
	int Dec = inner_data->thread_Dec;
	int nprime_total = inner_data->thread_nprime_total;
	Npp16sc *h_rawdata = inner_data->thread_h_rawdata;
	Npp16sc *d_in = inner_data->thread_d_in;
	Npp16sc *d_in_next = inner_data->thread_d_in_next;
	Npp32fc *d_out = inner_data->thread_d_out;
	Npp32fc *d_out_next = inner_data->thread_d_out_next;
	Npp32f *d_ftap = inner_data->thread_d_ftap;
	Npp32fc *h_out = inner_data->thread_h_out;
	hipStream_t stream = inner_data->thread_stream;
	hipStream_t copystream = inner_data->thread_copystream;
	int *nprime_startIdx = inner_data->thread_nprime_startIdx;
	
	Ipp8u *pDFTBuffer = inner_data->thread_pDFTBuffer;
	IppsDFTSpec_C_32fc *pDFTSpec = inner_data->thread_pDFTSpec;
	
	Ipp32fc *out = inner_data->thread_out; // for R2018
	// end of assignments
	
	hipError_t err;
	
    int k;
	Npp32fc *d_32fc_xfer;
	Npp16sc *d_16sc_xfer; // some pointers used for swapping
	int n, nprime, nprime_end, h_idx2copy;
	int nprime_start = nprime_startIdx[t_ID];
	if (t_ID == NUM_THREADS-1){ // if it's the last thread
		nprime_end = nprime_total; // the last index
	}
	else{
		nprime_end = nprime_startIdx[t_ID+1]; // otherwise just until before the next thread
	}
	
	for (nprime = nprime_start; nprime<nprime_end; nprime++){
		n = nprime * Dec;
		h_idx2copy = n + 1; 
		// now we start the async copies, except for the last iteration
		if(nprime != nprime_end-1){
			 hipMemcpyAsync(&d_in_next[0], &d_in[Dec], (L-Dec)*sizeof(Npp16sc), hipMemcpyDeviceToDevice, copystream); // here we move the array 'forward'
//			device_copy_vector4_int16_kernel<<<((L-Dec)/4 + 1024 - 1)/1024,1024,0,copystream>>>(&d_in_next[0], &d_in[Dec], (L-Dec)); // this appears to be ~5% faster, not sure if correct though
			hipMemcpyAsync(&d_in_next[L-Dec], &h_rawdata[h_idx2copy], Dec*sizeof(Npp16sc), hipMemcpyHostToDevice, copystream); // copy the new data at the end of the array
		}
		// while the copies are happening, do the computations
		
		// COMPUTATIONS
		wola_front<<<(N/1024)+1, 1024, 0, stream>>>(N, L, d_in, d_out_next, d_ftap); // you should just use N total threads, even if it's not concurrent!
		
		err = hipGetLastError();
		if (err != hipSuccess){
		    printf("Error: %s\n", hipGetErrorString(err));
		}

		hipStreamQuery(stream); // at this point, the memcpyasync from the previous iteration may not have finished yet..
		hipStreamSynchronize(stream); // DO NOT REMOVE THESE OR ELSE THE FFT MAY BE OPERATING ON OLD OUTPUT!
		if(nprime != nprime_start){ // the first iteration is all zeros, hasn't processed anything yet
			// === IPP DFT VERSION ===
			ippsDFTInv_CToC_32fc((Ipp32fc*)h_out, &out[(nprime-1)*N], pDFTSpec, pDFTBuffer);
			if (Dec*2 == N && (nprime-1) % 2 != 0){ // only if using overlapping channels, do some phase corrections when nprime is odd
                for (k=1; k<N; k=k+2){ //  all even k are definitely even in the product anyway
                    out[(nprime-1)*N + k].re = -out[(nprime-1)*N + k].re;
                    out[(nprime-1)*N + k].im = -out[(nprime-1)*N + k].im;
                }
            }
		}
		// END OF COMPUTATIONS
		
		d_32fc_xfer = d_out_next;
		d_out_next = d_out; // this frees up the array for the next iteration of processing in the kernel
		d_out = d_32fc_xfer; // now we can copy this out without having to worry that the kernel is going to work on the data being copied out
		hipMemcpyAsync(h_out, d_out, N*sizeof(Npp32fc), hipMemcpyDeviceToHost, stream); // we use the stream to copy out, copystream to copy in
		
		hipStreamQuery(copystream);
		hipStreamSynchronize(copystream); // make sure the early copies into the device are done first, then switch the waiting buffer
		d_16sc_xfer = d_in; // pointer swapping
		d_in = d_in_next; // now the 'current' data points to the waiting buffer
		d_in_next = d_16sc_xfer; // and finally the 'waiting' data points back to the first buffer
    }
	
	hipStreamQuery(stream); 
	hipStreamSynchronize(stream);
	// you need to process hout to fout one last time
	// === IPP DFT VERSION ===
	ippsDFTInv_CToC_32fc((Ipp32fc*)h_out, &out[(nprime_end-1)*N], pDFTSpec, pDFTBuffer);
	if (Dec*2 == N && (nprime_end-1) % 2 != 0){ // only if using overlapping channels, do some phase corrections when nprime is odd
		for (k=1; k<N; k=k+2){ //  all even k are definitely even in the product anyway
			out[(nprime_end-1)*N + k].re = -out[(nprime_end-1)*N + k].re;
			out[(nprime_end-1)*N + k].im = -out[(nprime_end-1)*N + k].im;
		}
	}
	
	_endthreadex(0); // only in windows
    return 0;
}

/* The gateway function */
void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
    ippInit();
	// == INITIALIZE TIMING ==
	int start_t = StartCounter();
	int end_t;
	// struct timespec vartime;
	double totalTime;
	
    size_t free_byte ; // these are for gpu checks
	size_t total_byte ;
    hipMemGetInfo( &free_byte, &total_byte );
    printf("Before alloc, GPU memory usage: free = %f MB, total = %f MB\n",(double)free_byte/1024.0/1024.0,(double)total_byte/1024.0/1024.0);
    
    // declare variables
    // Npp16s *y, *y_i; // pre R2018
	Npp16sc *y; // R2018
    Npp32f *h_ftap; // direct matlab inputs are always in doubles
	int nprimePts;
	int N, rawdataLength, L, Dec, DlyLen;
	// declare outputs
	// double *out_r, *out_i; // pre R2018
	mxComplexSingle *out; // for R2018
    
	
    //reserve stuff for windows threads
    int t; // for loops over threads
	HANDLE ThreadList[NUM_THREADS]; // handles to threads
    
    // // stuff for pthreads
    // pthread_t ThreadList[NUM_THREADS];
    // pthread_attr_t attr;
    // pthread_attr_init(&attr);
    // pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);
    
    /* check for proper number of arguments */
    if (nrhs!=4){
        mexErrMsgIdAndTxt("MyToolbox:arrayProduct:nrhs","4 Inputs required.");
    }

	// y = (Npp16s*)mxGetPr(prhs[0]); 
	// y_i = (Npp16s*)mxGetPi(prhs[0]);
	y = (Npp16sc*)mxGetComplexInt16s(prhs[0]);
    h_ftap = (Npp32f*)mxGetSingles(prhs[1]); 
	N = (int)mxGetScalar(prhs[2]); // this is the length of the fft i.e. number of channels
    Dec = (int)mxGetScalar(prhs[3]); // decimation factor
    
    rawdataLength = (int)(mxGetM(prhs[0]) * mxGetN(prhs[0])); // signal length
    L = (int)(mxGetM(prhs[1]) * mxGetN(prhs[1])); // no. of filter taps
    DlyLen = L - 1;
	
	/* argument checks */
    if (Dec != N && Dec*2 != N){
        mexErrMsgTxt("PHASE CORRECTION ONLY IMPLEMENTED FOR DECIMATION = FFT LENGTH OR DECIMATION * 2 = FFT LENGTH!");
    }
	if (L%N != 0){
        mexErrMsgTxt("Filter taps length must be factor multiple of fft length!");
    }
	
	nprimePts = (int)(rawdataLength/Dec);
	
	/* create the output matrix ===== TEST WITH COMPLEX DATA*/
    // plhs[0] = mxCreateDoubleMatrix(N,nprimePts,mxCOMPLEX);
	plhs[0] = mxCreateUninitNumericMatrix(N,nprimePts,mxSINGLE_CLASS,mxCOMPLEX);
    /* get a pointer to the real data in the output matrix */
    // out_r = mxGetPr(plhs[0]); 
    // out_i = mxGetPi(plhs[0]); // for pre R2018
	out = mxGetComplexSingles(plhs[0]);

    // declare extra pointer for consistency
    Npp16sc *h_rawdata;
    // // we make the interleaved input here 
    // hipHostMalloc((void**)&h_rawdata, sizeof(Npp16sc)*rawdataLength);
    // ippsRealToCplx_16s((Ipp16s*)y, (Ipp16s*)y_i, (Ipp16sc*)h_rawdata, rawdataLength);
    // or else, if the input is already interleaved for R2018 then
    h_rawdata = y;

    // cuda/fftw allocations
	Npp16sc *d_in, *d_in_next;
	Npp32f *d_ftap;
	Npp32fc *d_out, *d_out_next, *h_out;
	hipMalloc((void**)&d_ftap, sizeof(Npp32f)*L); // this is directly copying the entire ftap
	hipMalloc((void**)&d_in, sizeof(Npp16sc)*L*NUM_THREADS); // you will need at the most the same length as ftap
	hipMalloc((void**)&d_in_next, sizeof(Npp16sc)*L*NUM_THREADS); // we have a 2nd waiting buffer
	hipMalloc((void**)&d_out, sizeof(Npp32fc)*N*NUM_THREADS); // the output should be the N
	hipMalloc((void**)&d_out_next, sizeof(Npp32fc)*N*NUM_THREADS); // and a 2nd waiting buffer for output as well
	hipHostMalloc((void**)&h_out, sizeof(Npp32fc)*N*NUM_THREADS); // for transferring the output to the host
	hipMemcpy(d_ftap,h_ftap,L*sizeof(Npp32f), hipMemcpyHostToDevice);
	printf("Copied ftap to device \n");
	hipMemGetInfo( &free_byte, &total_byte );
	printf("After alloc, GPU memory usage: free = %f MB, total = %f MB\n",(double)free_byte/1024.0/1024.0,(double)total_byte/1024.0/1024.0);
    
	// === IPP DFT Allocs ===
	int sizeSpec = 0, sizeInit = 0, sizeBuf = 0;   
	ippsDFTGetSize_C_32fc(N, IPP_FFT_NODIV_BY_ANY, ippAlgHintNone, &sizeSpec, &sizeInit, &sizeBuf); // this just fills the 3 integers
	IppsDFTSpec_C_32fc **pDFTSpec = (IppsDFTSpec_C_32fc**)ippMalloc(sizeof(IppsDFTSpec_C_32fc*)*NUM_THREADS);
	Ipp8u **pDFTBuffer = (Ipp8u**)ippMalloc(sizeof(Ipp8u*)*NUM_THREADS);
	Ipp8u **pDFTMemInit = (Ipp8u**)ippMalloc(sizeof(Ipp8u*)*NUM_THREADS);
	
    // === multi-threaded approach ===
	hipStream_t streams[NUM_THREADS];
	hipStream_t copystreams[NUM_THREADS];
	
	int nprime_total = rawdataLength/Dec;
	int nprime_startIdx[NUM_THREADS];
	int n_start[NUM_THREADS];
	printf("i haven't crashed after allocating streams on stack and some nprime arrays\n");
    
    // we do ALL the initial copies first, so that we don't invoke invoke async copies later on in the threads
	start_t = GetCounter();
	for (t=0;t<NUM_THREADS;t++){
		nprime_startIdx[t] = nprime_total/NUM_THREADS * t;
		n_start[t] = nprime_startIdx[t] * Dec;
		if (n_start[t]<L){ // then you will be copying less than the full 500k but at the end of the destination array
			nppsZero_16sc(&d_in[t*L], L); // zero out the starting part, you'll have to run this before launching threads though? only can use 1 stream
			hipMemcpy(&d_in[t*L + DlyLen-n_start[t]],&h_rawdata[n_start[t]],(n_start[t]+1)*sizeof(Npp16sc), hipMemcpyHostToDevice); 
			printf("ZEROED: Initial copy of rawdata for thread %i done at n_start = %i, nprime_start = %i \n", t, n_start[t], nprime_startIdx[t]);
		}
		else{ // otherwise you copy the full 500k, but from an earlier part of the source array (to accommodate the delay for the filter)
			hipMemcpy(&d_in[t*L],&h_rawdata[n_start[t]-DlyLen],L*sizeof(Npp16sc),hipMemcpyHostToDevice);
			printf("Initial copy of rawdata for thread %i done at n_start = %i, nprime_start = %i \n", t, n_start[t], nprime_startIdx[t]);
		}
		// these copies should be correct
		
		hipStreamCreate(&streams[t]);
		hipStreamCreate(&copystreams[t]);
		printf("Streams created for thread %i \n", t);
		
		// ===== IPP DFT Initialization =====
		pDFTSpec[t] = (IppsDFTSpec_C_32fc*)ippMalloc(sizeSpec); // this is analogue of the fftw plan
		pDFTBuffer[t] = (Ipp8u*)ippMalloc(sizeBuf);
		pDFTMemInit[t] = (Ipp8u*)ippMalloc(sizeInit);
		ippsDFTInit_C_32fc(N, IPP_FFT_NODIV_BY_ANY, ippAlgHintNone,  pDFTSpec[t], pDFTMemInit[t]); // kinda like making the fftw plan?
	}
	end_t = GetCounter();
	totalTime = (end_t - start_t)/PCFreq; // in ms
	printf("Time for initial copies/stream/plan creation = %g ms \n",totalTime);
    
    // start threads
	start_t = GetCounter();
	for (t=0; t<NUM_THREADS; t++){
		thread_data_array[t].thread_t_ID = t;
		thread_data_array[t].thread_L = L;
		thread_data_array[t].thread_N = N;
		thread_data_array[t].thread_Dec = Dec;
		thread_data_array[t].thread_nprime_total = nprime_total;
		thread_data_array[t].thread_h_rawdata = h_rawdata;
		thread_data_array[t].thread_d_in = &d_in[t*L];
		thread_data_array[t].thread_d_in_next = &d_in_next[t*L];
		thread_data_array[t].thread_d_out = &d_out[t*N];
		thread_data_array[t].thread_d_out_next = &d_out_next[t*N];
		thread_data_array[t].thread_d_ftap = d_ftap;
		thread_data_array[t].thread_h_out = &h_out[t*N];
		thread_data_array[t].thread_stream = streams[t];
		thread_data_array[t].thread_copystream = copystreams[t];
		thread_data_array[t].thread_nprime_startIdx = nprime_startIdx;
		
		thread_data_array[t].thread_pDFTBuffer = pDFTBuffer[t];
		thread_data_array[t].thread_pDFTSpec = pDFTSpec[t];

		// thread_data_array[t].thread_out_r = (Ipp32f*)&out_r[0];
		// thread_data_array[t].thread_out_i = (Ipp32f*)&out_i[0]; // for pre R2018
		thread_data_array[t].thread_out = (Ipp32fc*)&out[0]; // for R2018
		
        // pthread_create(&ThreadList[t], &attr, threaded_wola, (void *)&thread_data_array[t]);
		ThreadList[t] = (HANDLE)_beginthreadex(NULL,0,&threaded_wola,(void*)&thread_data_array[t],0,NULL);

        printf("Beginning threadID %i..\n",thread_data_array[t].thread_t_ID);
	}
	
	WaitForMultipleObjects(NUM_THREADS,ThreadList,1,INFINITE);

	// ============== CLEANUP =================
	// close threads
	printf("Closing threads...\n");
	for(t=0;t<NUM_THREADS;t++){
	   CloseHandle(ThreadList[t]);
	//         printf("Closing threadID %i.. %i\n",(int)ThreadIDList[t],WaitForThread[t]);
	}
	printf("All threads closed! \n");
	end_t = GetCounter();
	totalTime = (end_t - start_t)/PCFreq; // in ms
	printf("Time for threads to finish = %g ms \n",totalTime);
	// =====================================

    for (t=0;t<NUM_THREADS;t++){
		hipStreamDestroy(streams[t]); 
		hipStreamDestroy(copystreams[t]);

		ippFree(pDFTSpec[t]);
		ippFree(pDFTBuffer[t]);
		ippFree(pDFTMemInit[t]);
	}
	ippFree(pDFTSpec);
	ippFree(pDFTBuffer);
	ippFree(pDFTMemInit);
	
    // hipHostFree(h_rawdata); // you only need this in older than R2018
	hipHostFree(h_out);
	hipFree(d_in); hipFree(d_in_next);
	hipFree(d_ftap);
	hipFree(d_out); hipFree(d_out_next);
	// do not free h_ftap! it's your input from matlab!
   
   // hipDeviceReset();
   // mexAtExit(cleanUp);
}
